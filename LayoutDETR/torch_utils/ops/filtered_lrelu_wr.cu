// Copyright (c) 2023 Salesforce, Inc.
// All rights reserved.
// SPDX-License-Identifier: Apache License 2.0
// For full license text, see LICENSE.txt file in the repo root or http://www.apache.org/licenses/
// By Ning Yu

// Redistributed from StyleGAN3 repo: https://github.com/NVlabs/stylegan3
// Copyright (c) 2021, NVIDIA CORPORATION & AFFILIATES.  All rights reserved.

#include "filtered_lrelu.cu"

// Template/kernel specializations for sign write mode.

// Full op, 32-bit indexing.
template filtered_lrelu_kernel_spec choose_filtered_lrelu_kernel<c10::Half, int32_t, true, false>(const filtered_lrelu_kernel_params& p, int sharedKB);
template filtered_lrelu_kernel_spec choose_filtered_lrelu_kernel<float,     int32_t, true, false>(const filtered_lrelu_kernel_params& p, int sharedKB);

// Full op, 64-bit indexing.
template filtered_lrelu_kernel_spec choose_filtered_lrelu_kernel<c10::Half, int64_t, true, false>(const filtered_lrelu_kernel_params& p, int sharedKB);
template filtered_lrelu_kernel_spec choose_filtered_lrelu_kernel<float,     int64_t, true, false>(const filtered_lrelu_kernel_params& p, int sharedKB);

// Activation/signs only for generic variant. 64-bit indexing.
template void* choose_filtered_lrelu_act_kernel<c10::Half, true, false>(void);
template void* choose_filtered_lrelu_act_kernel<float,     true, false>(void);
template void* choose_filtered_lrelu_act_kernel<double,    true, false>(void);

// Copy filters to constant memory.
template hipError_t copy_filters<true, false>(hipStream_t stream);
